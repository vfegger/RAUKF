#include "hip/hip_runtime.h"
#include "../include/hc2D.hpp"

__host__ __device__ inline double HC2D::C(double x, double y)
{
    return 385 * 8960;
}
__host__ __device__ inline double HC2D::K(double x, double y)
{
    return 400;
}

void HC2D::validate(HCParms &parms)
{
    if (!(refparms == parms))
    {
        refparms = parms;
        if (AI.host())
        {
            AI.free();
        }
        int Lxy = parms.Lx * parms.Ly;
        int Lu = 1 + 2 * (parms.Lx + parms.Ly);
        int L = Lxy + Lxy;
        int L2 = L * L;
        AI.alloc(L2);
        BE.alloc(L2);
        CE.alloc(L * Lu);
        ATA.alloc(L2);
        JX.alloc(L2);
        JU.alloc(L * Lu);
        isValid = false;
    }
}

void HC2D::CPU::ImplicitScheme(HCParms &parms, int strideTQ, int strideAC)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1 + 2 * (parms.Lx + parms.Ly);
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;
    double gamma = parms.gamma;

    double *pAI = AI.host();
    double *pBE = BE.host();
    double *pCE = CE.host();
    double *pATA = ATA.host();
    double *paux = (double *)malloc(sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pUT, *pUQ;
    MathCPU::Zero(pAI, L * L);
    MathCPU::Identity(pBE, L, L);
    MathCPU::Zero(pCE, L * Lu);
    MathCPU::Zero(pATA, L * L);
    MathCPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pUT = pCE + std::max(0, -strideTQ);
    pUQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.host();
    double *JUh = JU.host();
    // Difusion Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            double aux = 0.0;
            if (i != 0)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] = -dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
            }
            if (i != Lx - 1)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] = -dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
            }
            if (j != 0)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] = -dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
                pTT[(index + 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
            }
            if (j != Ly - 1)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] = -dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
                pTT[(index - 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
            }
            pTT[index * L + index] = 1.0 + aux + dt * h / (c * CT);
        }
    }

    // Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQT[index * L + index] = -dt * amp / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQQ[index * L + index] = 1.0;
        }
    }

    // Control Input Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pUT[index] = h / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        double KT = K((0.5) * dx, (j + 0.5) * dy);
        double CT = C((0.5) * dx, (j + 0.5) * dy);
        int index_0 = j * Lx + 0;
        int index_1 = j * Lx + Lx - 1;
        pUT[(1 + j) * L + index_0] = -dt * gamma * KT / (CT * dx * dx);
        pUT[(1 + Ly + j) * L + index_1] = -dt * gamma * KT / (CT * dx * dx);
    }
    for (int i = 0; i < Lx; ++i)
    {
        double KT = K((i + 0.5) * dx, (0.5) * dy);
        double CT = C((i + 0.5) * dx, (0.5) * dy);
        int index_0 = 0 * Lx + i;
        int index_1 = (Ly - 1) * Lx + i;
        pUT[(1 + 2 * Ly + i) * L + index_0] = -dt * gamma * KT / (CT * dx * dx);
        pUT[(1 + 2 * Ly + Lx + i) * L + index_1] = -dt * gamma * KT / (CT * dx * dx);
    }
    // Control Input Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            index = j * Lx + i;
            pUQ[index] = 0.0;
        }
    }

    MathCPU::MatMulTN(0.0, pATA, 1.0, pAI, pAI, L, L, L);
    MathCPU::MatMulTN(0.0, paux, 1.0, pAI, pBE, L, L, L);
    // Solve JX = (A^T * A)^-1 * A^T * B
    MathCPU::CholeskySolver(JXh, pATA, paux, L, L, L);

    MathCPU::MatMulTN(0.0, paux, 1.0, pAI, pCE, L, L, Lu);
    // Solve JU = (A^T * A)^-1 * A^T * C
    MathCPU::CholeskySolver(JUh, pATA, paux, L, L, Lu);
    free(paux);
}

void HC2D::CPU::ExplicitScheme(HCParms &parms, int strideTQ, int strideAC)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1 + 2 * (parms.Lx + parms.Ly);
    ;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;

    double *pAI = AI.host();
    double *pBE = BE.host();
    double *pCE = CE.host();
    double *pATA = ATA.host();
    double *paux = (double *)malloc(sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pUT, *pUQ;
    MathCPU::Identity(pAI, L, L);
    MathCPU::Zero(pBE, L * L);
    MathCPU::Zero(pCE, L * Lu);
    MathCPU::Zero(pATA, L * L);
    MathCPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pUT = pCE + std::max(0, -strideTQ);
    pUQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.host();
    double *JUh = JU.host();
    // Difusion Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            double aux = 0.0;
            if (i != 0)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] = dt * KT / (CT * dx * dx);
            }
            else
            {
            }
            if (i != Lx - 1)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] = dt * KT / (CT * dx * dx);
            }
            else
            {
            }
            if (j != 0)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] = dt * KT / (CT * dy * dy);
            }
            else
            {
            }
            if (j != Ly - 1)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] = dt * KT / (CT * dy * dy);
            }
            else
            {
            }
            pTT[index * L + index] = 1.0 - aux - dt * h / (c * CT);
        }
    }

    // Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQT[index * L + index] = dt * amp / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQQ[index * L + index] = 1.0;
        }
    }

    // Control Input Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pUT[index] = h / (c * CT);
        }
    }
    // Control Input Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            index = j * Lx + i;
            pUQ[index] = 0.0;
        }
    }

    // Solve JX = (A^T * A)^-1 * A^T * B
    MathCPU::Copy(JXh, pBE, L * L);

    // Solve JU = (A^T * A)^-1 * A^T * C
    MathCPU::Copy(JUh, pCE, L * Lu);
}

void HC2D::CPU::EvolutionMatrix(HCParms &parms, double *pmXX_o, double *pmUX_o, int strideTQ, int strideAC)
{
    validate(parms);
    if (!isValid)
    {
#if IMPLICIT_SCHEME == 1
        ImplicitScheme(parms, strideTQ);
#else
        ExplicitScheme(parms, strideTQ);
#endif
        isValid = true;
    }
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    MathCPU::Copy(pmXX_o, JX.host(), L * L);
    MathCPU::Copy(pmUX_o, JU.host(), L * Lu);
}

void HC2D::CPU::EvaluationMatrix(HCParms &parms, double *pmH_o, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    double dx = parms.dx;
    double dy = parms.dy;
    double c = parms.Sz;
    double amp = parms.amp;

    double *pmTT, *pmQT;
    pmTT = pmH_o + std::max(-strideTQ, 0) * Lxy;
    pmQT = pmH_o + std::max(strideTQ, 0) * Lxy;
    // Surface Temperature
    for (int j = 0; j < Ly; j++)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pmTT[index * Lxy + index] = 1.0;
#if ILSA == 1
            pmQT[index * Lxy + index] = -c * amp / (6.0 * KT);
#endif
        }
    }
}

__global__ void ImplicitScheme_A(double *pmTT, double *pmTQ, double *pmQT, double *pmQQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        double aux = 0.0;
        if (i != 0)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] = -dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (i != Lx - 1)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] = -dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (j != 0)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] = -dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
        }
        if (j != Ly - 1)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] = -dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += -(1.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
        }
        pmTT[index * L + index] = 1.0 + aux + dt * h / (c * CT);
        pmQT[index * L + index] = -dt * amp / (c * CT);
        pmQQ[index * L + index] = 1.0;
    }
}

__global__ void ImplicitScheme_C(double *pmUT, double *pmUQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        pmUT[index] = dt * h / (c * CT);
        pmUQ[index] = 0.0;
        if (i == 0)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmUT[(k + 1) * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (i == Lx - 1)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmUT[(k + Ly + 1) * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (j == 0)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmUT[(k + 2 * Ly + 1) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
        if (j == Ly - 1)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmUT[(k + Lx + 2 * Ly + 1) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
    }
}

__global__ void ExplicitScheme_B(double *pmTT, double *pmTQ, double *pmQT, double *pmQQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        double aux = 0.0;
        if (i != 0)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] = dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += (1.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (i != Lx - 1)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] = dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += (1.0 / 4.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (j != 0)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] = dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += (1.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
        }
        if (j != Ly - 1)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] = dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (5.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += (1.0 / 4.0) * dt * gamma * KT / (CT * dy * dy);
        }
        pmTT[index * L + index] = 1.0 - aux - dt * h / (c * CT);
        pmQT[index * L + index] = dt * amp / (c * CT);
        pmQQ[index * L + index] = 1.0;
    }
}

__global__ void ExplicitScheme_C(double *pmUT, double *pmUQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        pmUT[index] = dt * h / (c * CT);
        pmUQ[index] = 0.0;
        if (i == 0)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmUT[(k + 1) * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (i == Lx - 1)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmUT[(k + Ly + 1) * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (j == 0)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmUT[(k + 2 * Ly + 1) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
        if (j == Ly - 1)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmUT[(k + Lx + 2 * Ly + 1) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
    }
}

void HC2D::GPU::ImplicitScheme(HCParms &parms, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;

    double *pAI = AI.dev();
    double *pBE = BE.dev();
    double *pCE = CE.dev();
    double *pATA = ATA.dev();
    double *paux = NULL;
    hipMalloc(&paux, sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pUT, *pUQ;
    MathGPU::Zero(pAI, L * L);
    MathGPU::Identity(pBE, L, L);
    MathGPU::Zero(pCE, L * Lu);
    MathGPU::Zero(pATA, L * L);
    MathGPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pUT = pCE + std::max(0, -strideTQ);
    pUQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.dev();
    double *JUh = JU.dev();

    dim3 T(16, 16);
    dim3 B((L + T.x - 1) / T.x, (L + T.y - 1) / T.y);
    ImplicitScheme_A<<<B, T>>>(pTT, pTQ, pQT, pQQ, Lx, Ly, dx, dy, dt, c, amp, h);
    MathGPU::Identity(pBE, L, L);
    ImplicitScheme_C<<<B, T>>>(pUT, pUQ, Lx, Ly, dx, dy, dt, c, amp, h);

    MathGPU::MatMulTN(0.0, pATA, 1.0, pAI, pAI, L, L, L);
    MathGPU::MatMulTN(0.0, paux, 1.0, pAI, pBE, L, L, L);
    // Solve JX = (A^T * A)^-1 * A^T * B
    MathGPU::CholeskySolver(JXh, pATA, paux, L, L, L);

    MathGPU::MatMulTN(0.0, paux, 1.0, pAI, pCE, L, L, Lu);
    // Solve JU = (A^T * A)^-1 * A^T * C
    MathGPU::CholeskySolver(JUh, pATA, paux, L, L, Lu);
    hipFree(paux);
}

void HC2D::GPU::ExplicitScheme(HCParms &parms, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;
    double gamma = parms.gamma;

    double *pAI = AI.dev();
    double *pBE = BE.dev();
    double *pCE = CE.dev();
    double *pATA = ATA.dev();
    double *paux = NULL;
    hipMalloc(&paux, sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pUT, *pUQ;
    MathGPU::Identity(pAI, L, L);
    MathGPU::Zero(pBE, L * L);
    MathGPU::Zero(pCE, L * Lu);
    MathGPU::Zero(pATA, L * L);
    MathGPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pUT = pCE + std::max(0, -strideTQ);
    pUQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.dev();
    double *JUh = JU.dev();

    dim3 T(16, 16);
    dim3 B((L + T.x - 1) / T.x, (L + T.y - 1) / T.y);
    ImplicitScheme_A<<<B, T>>>(pTT, pTQ, pQT, pQQ, Lx, Ly, dx, dy, dt, c, amp, h, gamma);
    MathGPU::Identity(pBE, L, L);
    ImplicitScheme_C<<<B, T>>>(pUT, pUQ, Lx, Ly, dx, dy, dt, c, amp, h, gamma);

    // Solve JX = (A^T * A)^-1 * A^T * B
    MathGPU::Copy(JXh, pBE, L * L);

    // Solve JU = (A^T * A)^-1 * A^T * C
    MathGPU::Copy(JUh, pCE, L * Lu);
}

void HC2D::GPU::EvolutionMatrix(HCParms &parms, double *pmXX_o, double *pmUX_o, int strideTQ)
{
    validate(parms);
    if (!isValid)
    {
#if IMPLICIT_SCHEME == 1
        ImplicitScheme(parms, strideTQ);
#else
        ExplicitScheme(parms, strideTQ);
#endif
        std::cout << "Saving matrices for Evolution" << std::endl;
        isValid = true;
    }
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    MathGPU::Copy(pmXX_o, JX.dev(), L * L);
    MathGPU::Copy(pmUX_o, JU.dev(), L * Lu);
}

void HC2D::GPU::EvaluationMatrix(HCParms &parms, double *pmH_o, int strideTQ)
{
    int Lxy = parms.Lx * parms.Ly;

    double *pm;

    pm = (double *)malloc(sizeof(double) * 2 * Lxy * Lxy);
    for (int i = 0; i < 2 * Lxy * Lxy; i++)
    {
        pm[i] = 0.0;
    }

    HC2D::CPU::EvaluationMatrix(parms, pm, strideTQ);

    hipMemcpy(pmH_o, pm, 2 * sizeof(double) * Lxy * Lxy, hipMemcpyKind::hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    free(pm);
}