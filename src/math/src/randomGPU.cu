#include "../include/randomGPU.hpp"

void RandomGPU::SampleNormal(double *v_o, int length, double mean, double sigma)
{
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetStream(generator,hipStreamDefault);
    hiprandGenerateNormalDouble(generator,v_o,length,mean,sigma);
    hiprandDestroyGenerator(generator);
}