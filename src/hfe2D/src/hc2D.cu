#include "hip/hip_runtime.h"
#include "../include/hc2D.hpp"

__host__ __device__ inline double HC2D::C(double x, double y)
{
    return 385 * 8960;
}
__host__ __device__ inline double HC2D::K(double x, double y)
{
    return 400;
}

void HC2D::validate(HCParms &parms)
{
    if (!(refparms == parms))
    {
        refparms = parms;
        if (AI.host())
        {
            AI.free();
        }
        int Lxy = parms.Lx * parms.Ly;
        int Lu = 1;
        int L = Lxy + Lxy;
        int L2 = L * L;
        AI.alloc(L2);
        BE.alloc(L2);
        CE.alloc(L * Lu);
        ATA.alloc(L2);
        JX.alloc(L2);
        JU.alloc(L * Lu);
        isValid = false;
    }
}

void HC2D::CPU::ImplicitScheme(HCParms &parms, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;

    double *pAI = AI.host();
    double *pBE = BE.host();
    double *pCE = CE.host();
    double *pATA = ATA.host();
    double *paux = (double *)malloc(sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ;
    MathCPU::Zero(pAI, L * L);
    MathCPU::Identity(pBE, L, L);
    MathCPU::Zero(pCE, L * Lu);
    MathCPU::Zero(pATA, L * L);
    MathCPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.host();
    double *JUh = JU.host();
    // Difusion Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            double aux = 0.0;
            if (i != 0)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] = -dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += +2 * dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] += -3 * dt * KT / (CT * dx * dx);
                pTT[(index + 2) * L + index] += +1 * dt * KT / (CT * dx * dx);
            }
            if (i != Lx - 1)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] = -dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += +2 * dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] += -3 * dt * KT / (CT * dx * dx);
                pTT[(index - 2) * L + index] += +1 * dt * KT / (CT * dx * dx);
            }
            if (j != 0)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] = -dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += +2 * dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] += -3 * dt * KT / (CT * dy * dy);
                pTT[(index + 2 * Lx) * L + index] += +1 * dt * KT / (CT * dy * dy);
            }
            if (j != Ly - 1)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] = -dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += +2 * dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] += -3 * dt * KT / (CT * dy * dy);
                pTT[(index - 2 * Lx) * L + index] += +1 * dt * KT / (CT * dy * dy);
            }
            pTT[index * L + index] = 1.0 + aux + dt * h / (c * CT);
        }
    }

    // Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQT[index * L + index] = -dt * amp / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQQ[index * L + index] = 1.0;
        }
    }

    // Ambient Temperature Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pTaT[index] = h / (c * CT);
        }
    } // Ambient Temperature Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            index = j * Lx + i;
            pTaQ[index] = 0.0;
        }
    }

    MathCPU::MatMulTN(0.0, pATA, 1.0, pAI, pAI, L, L, L);
    MathCPU::MatMulTN(0.0, paux, 1.0, pAI, pBE, L, L, L);
    // Solve JX = (A^T * A)^-1 * A^T * B
    MathCPU::CholeskySolver(JXh, pATA, paux, L, L, L);

    MathCPU::MatMulTN(0.0, paux, 1.0, pAI, pCE, L, L, Lu);
    // Solve JU = (A^T * A)^-1 * A^T * C
    MathCPU::CholeskySolver(JUh, pATA, paux, L, L, Lu);
    free(paux);
}

void HC2D::CPU::ExplicitScheme(HCParms &parms, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;

    double *pAI = AI.host();
    double *pBE = BE.host();
    double *pCE = CE.host();
    double *pATA = ATA.host();
    double *paux = (double *)malloc(sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ;
    MathCPU::Identity(pAI, L, L);
    MathCPU::Zero(pBE, L * L);
    MathCPU::Zero(pCE, L * Lu);
    MathCPU::Zero(pATA, L * L);
    MathCPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.host();
    double *JUh = JU.host();
    // Difusion Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            double aux = 0.0;
            if (i != 0)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] = dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += 2 * dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] += +3 * dt * KT / (CT * dx * dx);
                pTT[(index + 2) * L + index] += -1 * dt * KT / (CT * dx * dx);
            }
            if (i != Lx - 1)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] = dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += 2 * dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] += +3 * dt * KT / (CT * dx * dx);
                pTT[(index - 2) * L + index] += -1 * dt * KT / (CT * dx * dx);
            }
            if (j != 0)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] = dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += 2 * dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] += +3 * dt * KT / (CT * dy * dy);
                pTT[(index + 2 * Lx) * L + index] += -1 * dt * KT / (CT * dy * dy);
            }
            if (j != Ly - 1)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] = dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += 2 * dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] += +3 * dt * KT / (CT * dy * dy);
                pTT[(index - 2 * Lx) * L + index] += -1 * dt * KT / (CT * dy * dy);
            }
            pTT[index * L + index] = 1.0 - aux - dt * h / (c * CT);
        }
    }

    // Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQT[index * L + index] = dt * amp / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQQ[index * L + index] = 1.0;
        }
    }

    // Ambient Temperature Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pTaT[index] = h / (c * CT);
        }
    }
    // Ambient Temperature Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            index = j * Lx + i;
            pTaQ[index] = 0.0;
        }
    }

    // Solve JX = (A^T * A)^-1 * A^T * B
    MathCPU::Copy(JXh, pBE, L * L);

    // Solve JU = (A^T * A)^-1 * A^T * C
    MathCPU::Copy(JUh, pCE, L * Lu);
}

void HC2D::CPU::EvolutionMatrix(HCParms &parms, double *pmXX_o, double *pmUX_o, int strideTQ)
{
    validate(parms);
    if (!isValid)
    {
#if IMPLICIT_SCHEME == 1
        ImplicitScheme(parms, strideTQ);
#else
        ExplicitScheme(parms, strideTQ);
#endif
        isValid = true;
    }
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    MathCPU::Copy(pmXX_o, JX.host(), L * L);
    MathCPU::Copy(pmUX_o, JU.host(), L * Lu);
}

void HC2D::CPU::EvaluationMatrix(HCParms &parms, double *pmH_o, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    double dx = parms.dx;
    double dy = parms.dy;
#if ILSA == 1
    double c = parms.Sz;
    double amp = parms.amp;
#endif

    double *pmTT, *pmQT;
    pmTT = pmH_o + std::max(-strideTQ, 0) * Lxy;
    pmQT = pmH_o + std::max(strideTQ, 0) * Lxy;
    // Surface Temperature
    for (int j = 0; j < Ly; j++)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pmTT[index * Lxy + index] = 1.0;
#if ILSA == 1
            pmQT[index * Lxy + index] = -c * amp / (6.0 * KT);
#endif
        }
    }
}

__global__ void ImplicitScheme_A(double *pmTT, double *pmTQ, double *pmQT, double *pmQQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        double aux = 0.0;
        if (i != 0)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] += -dt * KT / (CT * dx * dx);
        }
        else
        {
#if ORDER == 1
            aux += -1 * dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += +1 * dt * KT / (CT * dx * dx);
#elif ORDER == 2
            aux += -2 * dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += +3 * dt * KT / (CT * dx * dx);
            pmTT[(index + 2) * L + index] += -1 * dt * KT / (CT * dx * dx);
#endif
        }
        if (i != Lx - 1)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += -dt * KT / (CT * dx * dx);
        }
        else
        {
#if ORDER == 0
#elif ORDER == 1
            aux += -1 * dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] += +1 * dt * KT / (CT * dx * dx);
#elif ORDER == 2
            aux += -2 * dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] += +3 * dt * KT / (CT * dx * dx);
            pmTT[(index - 2) * L + index] += -1 * dt * KT / (CT * dx * dx);
#endif
        }
        if (j != 0)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] += -dt * KT / (CT * dy * dy);
        }
        else
        {
#if ORDER == 0
#elif ORDER == 1
            aux += -1 * dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] += +1 * dt * KT / (CT * dy * dy);
#elif ORDER == 2
            aux += -2 * dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] += +3 * dt * KT / (CT * dy * dy);
            pmTT[(index + 2 * Lx) * L + index] += -1 * dt * KT / (CT * dy * dy);
#endif
        }
        if (j != Ly - 1)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] += -dt * KT / (CT * dy * dy);
        }
        else
        {
#if ORDER == 0
#elif ORDER == 1
            aux += -1 * dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] += +1 * dt * KT / (CT * dy * dy);
#elif ORDER == 2
            aux += -2 * dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] += +3 * dt * KT / (CT * dy * dy);
            pmTT[(index - 2 * Lx) * L + index] += -1 * dt * KT / (CT * dy * dy);
#endif
        }
        pmTT[index * L + index] = 1.0 + aux + dt * h / (c * CT);
        pmQT[index * L + index] = -dt * amp / (c * CT);
        pmQQ[index * L + index] = 1.0;
    }
}

__global__ void ImplicitScheme_C(double *pmTaT, double *pmTaQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        pmTaT[index] = dt * h / (c * CT);
        pmTaQ[index] = 0.0;
    }
}

__global__ void ExplicitScheme_B(double *pmTT, double *pmTQ, double *pmQT, double *pmQQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        double aux = 0.0;
        if (i != 0)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] = dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += 2 * dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += 3 * dt * KT / (CT * dx * dx);
            pmTT[(index + 2) * L + index] += -1 * dt * KT / (CT * dx * dx);
        }
        if (i != Lx - 1)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] = dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += 2 * dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] += +3 * dt * KT / (CT * dx * dx);
            pmTT[(index - 2) * L + index] += -1 * dt * KT / (CT * dx * dx);
        }
        if (j != 0)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] = dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += 2 * dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] += +3 * dt * KT / (CT * dy * dy);
            pmTT[(index + 2 * Lx) * L + index] += -1 * dt * KT / (CT * dy * dy);
        }
        if (j != Ly - 1)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] = dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += 2 * dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] += 3 * dt * KT / (CT * dy * dy);
            pmTT[(index - 2 * Lx) * L + index] += -1 * dt * KT / (CT * dy * dy);
        }
        pmTT[index * L + index] = 1.0 - aux - dt * h / (c * CT);
        pmQT[index * L + index] = dt * amp / (c * CT);
        pmQQ[index * L + index] = 1.0;
    }
}

__global__ void ExplicitScheme_C(double *pmTaT, double *pmTaQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        pmTaT[index] = dt * h / (c * CT);
        pmTaQ[index] = 0.0;
    }
}

void HC2D::GPU::ImplicitScheme(HCParms &parms, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;

    double *pAI = AI.dev();
    double *pBE = BE.dev();
    double *pCE = CE.dev();
    double *pATA = ATA.dev();
    double *paux = NULL;
    hipMalloc(&paux, sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ;
    MathGPU::Zero(pAI, L * L);
    MathGPU::Identity(pBE, L, L);
    MathGPU::Zero(pCE, L * Lu);
    MathGPU::Zero(pATA, L * L);
    MathGPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, strideTQ);

    double *JXd = JX.dev();
    double *JUd = JU.dev();

    dim3 T(16, 16);
    dim3 B((L + T.x - 1) / T.x, (L + T.y - 1) / T.y);
    ImplicitScheme_A<<<B, T>>>(pTT, pTQ, pQT, pQQ, Lx, Ly, dx, dy, dt, c, amp, h);
    MathGPU::Identity(pBE, L, L);
    ImplicitScheme_C<<<B, T>>>(pTaT, pTaQ, Lx, Ly, dx, dy, dt, c, amp, h);

    MathGPU::MatMulTN(0.0, pATA, 1.0, pAI, pAI, L, L, L);
    MathGPU::MatMulTN(0.0, paux, 1.0, pAI, pBE, L, L, L);
    // Solve JX = (A^T * A)^-1 * A^T * B
    MathGPU::CholeskySolver(JXd, pATA, paux, L, L, L);

    MathGPU::MatMulTN(0.0, paux, 1.0, pAI, pCE, L, L, Lu);
    // Solve JU = (A^T * A)^-1 * A^T * C
    MathGPU::CholeskySolver(JUd, pATA, paux, L, L, Lu);
    hipFree(paux);
}

void HC2D::GPU::ExplicitScheme(HCParms &parms, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;

    double *pAI = AI.dev();
    double *pBE = BE.dev();
    double *pCE = CE.dev();
    double *pATA = ATA.dev();
    double *paux = NULL;
    hipMalloc(&paux, sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ;
    MathGPU::Identity(pAI, L, L);
    MathGPU::Zero(pBE, L * L);
    MathGPU::Zero(pCE, L * Lu);
    MathGPU::Zero(pATA, L * L);
    MathGPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, strideTQ);

    double *JXh = JX.dev();
    double *JUh = JU.dev();

    dim3 T(16, 16);
    dim3 B((L + T.x - 1) / T.x, (L + T.y - 1) / T.y);
    ImplicitScheme_A<<<B, T>>>(pTT, pTQ, pQT, pQQ, Lx, Ly, dx, dy, dt, c, amp, h);
    MathGPU::Identity(pBE, L, L);
    ImplicitScheme_C<<<B, T>>>(pTaT, pTaQ, Lx, Ly, dx, dy, dt, c, amp, h);

    // Solve JX = (A^T * A)^-1 * A^T * B
    MathGPU::Copy(JXh, pBE, L * L);

    // Solve JU = (A^T * A)^-1 * A^T * C
    MathGPU::Copy(JUh, pCE, L * Lu);
}

void HC2D::GPU::EvolutionMatrix(HCParms &parms, double *pmXX_o, double *pmUX_o, int strideTQ)
{
    validate(parms);
    if (!isValid)
    {
#if IMPLICIT_SCHEME == 1
        ImplicitScheme(parms, strideTQ);
#else
        ExplicitScheme(parms, strideTQ);
#endif
        std::cout << "Saving matrices for Evolution" << std::endl;
        isValid = true;
    }
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    MathGPU::Copy(pmXX_o, JX.dev(), L * L);
    MathGPU::Copy(pmUX_o, JU.dev(), L * Lu);
}

void HC2D::GPU::EvaluationMatrix(HCParms &parms, double *pmH_o, int strideTQ)
{
    int Lxy = parms.Lx * parms.Ly;

    double *pm;

    pm = (double *)malloc(sizeof(double) * 2 * Lxy * Lxy);
    for (int i = 0; i < 2 * Lxy * Lxy; i++)
    {
        pm[i] = 0.0;
    }

    HC2D::CPU::EvaluationMatrix(parms, pm, strideTQ);

    hipMemcpy(pmH_o, pm, 2 * sizeof(double) * Lxy * Lxy, hipMemcpyKind::hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    free(pm);
}