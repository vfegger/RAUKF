#include "hip/hip_runtime.h"
#include "../include/mathGPU.hpp"

__global__ void CUDA_Add(double *pv_io, double *pv_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_io[index] += pv_i[index];
    }
}
__global__ void CUDA_Sub(double *pv_io, double *pv_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_io[index] -= pv_i[index];
    }
}
__global__ void CUDA_Mul(double *pv_io, double v_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_io[index] *= v_i;
    }
}
__global__ void CUDA_Mul(double *pv_io, double *pv_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_io[index] *= pv_i[index];
    }
}

__global__ void CUDA_Add(double *pv_o, double *pvL_i, double *pvR_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_o[index] = pvL_i[index] + pvR_i[index];
    }
}
__global__ void CUDA_Sub(double *pv_o, double *pvL_i, double *pvR_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_o[index] = pvL_i[index] - pvR_i[index];
    }
}
__global__ void CUDA_Mul(double *pv_o, double *pvL_i, double vR_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_o[index] = pvL_i[index] * vR_i;
    }
}
__global__ void CUDA_Mul(double *pv_o, double *pvL_i, double *pvR_i, unsigned int length)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < length)
    {
        pv_o[index] = pvL_i[index] * pvR_i[index];
    }
}

__global__ void CUDA_Mean(double *pv_o, double *pm_i, unsigned int lengthI, unsigned int lengthJ)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < lengthI)
    {
        double acc = 0.0;
        for (unsigned j = 0; j < lengthJ; ++j)
        {
            acc += pm_i[j * lengthI + index];
        }
        pv_o[index] = acc / lengthJ;
    }
}

__global__ void CUDA_Mean(double *pv_o, double *pm_i, double *pw_i, unsigned int lengthI, unsigned int lengthJ)
{
    unsigned index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < lengthI)
    {
        double acc = 0.0;
        for (unsigned j = 0; j < lengthJ; ++j)
        {
            acc += pw_i[j] * pm_i[j * lengthI + index];
        }
        pv_o[index] = acc;
    }
}

void MathGPU::CreateHandles()
{
    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverDnHandle);
}
void MathGPU::DestroyHandles()
{
    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverDnHandle);
}

void MathGPU::Zero(double *pv_o, int length)
{
    hipMemsetAsync(pv_o, 0, sizeof(double) * length, hipStreamDefault);
}
void MathGPU::Copy(double *pv_o, double *pv_i, int length)
{
    hipMemcpyAsync(pv_o, pv_i, sizeof(double) * length, hipMemcpyKind::hipMemcpyDeviceToDevice, hipStreamDefault);
}

void MathGPU::Add(double *pv_io, double *pv_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Add<<<B, T, 0, hipStreamDefault>>>(pv_io, pv_i, length);
}
void MathGPU::Sub(double *pv_io, double *pv_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Sub<<<B, T, 0, hipStreamDefault>>>(pv_io, pv_i, length);
}
void MathGPU::Mul(double *pv_io, double v_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Mul<<<B, T, 0, hipStreamDefault>>>(pv_io, v_i, length);
}
void MathGPU::Mul(double *pv_io, double *pv_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Mul<<<B, T, 0, hipStreamDefault>>>(pv_io, pv_i, length);
}
void MathGPU::Add(double *pv_o, double *pvL_i, double *pvR_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Add<<<B, T, 0, hipStreamDefault>>>(pv_o, pvL_i, pvR_i, length);
}
void MathGPU::Sub(double *pv_o, double *pvL_i, double *pvR_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Sub<<<B, T, 0, hipStreamDefault>>>(pv_o, pvL_i, pvR_i, length);
}
void MathGPU::Mul(double *pv_o, double *pvL_i, double vR_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Mul<<<B, T, 0, hipStreamDefault>>>(pv_o, pvL_i, vR_i, length);
}
void MathGPU::Mul(double *pv_o, double *pvL_i, double *pvR_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    CUDA_Mul<<<B, T, 0, hipStreamDefault>>>(pv_o, pvL_i, pvR_i, length);
}
void MathGPU::MatMulNN(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, int M, int K, int N)
{
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, pmL_i, M, pmR_i, K, &beta, pm_o, M);
}
void MathGPU::MatMulNWN(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, double *pw_i, int M, int K, int N)
{
    double *aux;
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipMallocAsync(&aux, sizeof(double) * min(M, N) * K, hipStreamDefault);
    Zero(aux, min(M, N) * K);
    if (M < N)
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_RIGHT, M, K, pmL_i, M, pw_i, 1, aux, M);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, aux, M, pmR_i, K, &beta, pm_o, M);
    }
    else
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_LEFT, K, N, pmR_i, K, pw_i, 1, aux, K);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, pmL_i, M, aux, K, &beta, pm_o, M);
    }
    hipFreeAsync(aux, hipStreamDefault);
}
void MathGPU::MatMulNT(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, int M, int K, int N)
{
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T, M, N, K, &alpha, pmL_i, M, pmR_i, N, &beta, pm_o, M);
}
void MathGPU::MatMulNWT(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, double *pw_i, int M, int K, int N)
{
    double *aux;
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipMallocAsync(&aux, sizeof(double) * min(M, N) * K, hipStreamDefault);
    Zero(aux, min(M, N) * K);
    if (M < N)
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_RIGHT, M, K, pmL_i, M, pw_i, 1, aux, M);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T, M, N, K, &alpha, aux, M, pmR_i, N, &beta, pm_o, M);
    }
    else
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_RIGHT, N, K, pmR_i, M, pw_i, 1, aux, M);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T, M, N, K, &alpha, pmL_i, M, aux, N, &beta, pm_o, M);
    }
    hipFreeAsync(aux, hipStreamDefault);
}
void MathGPU::MatMulTN(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, int M, int K, int N)
{
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, pmL_i, K, pmR_i, K, &beta, pm_o, M);
}
void MathGPU::MatMulTWN(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, double *pw_i, int M, int K, int N)
{
    double *aux;
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipMallocAsync(&aux, sizeof(double) * min(M, N) * K, hipStreamDefault);
    Zero(aux, min(M, N) * K);
    if (M < N)
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_LEFT, K, M, pmL_i, K, pw_i, 1, aux, K);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, aux, K, pmR_i, K, &beta, pm_o, M);
    }
    else
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_LEFT, K, N, pmR_i, K, pw_i, 1, aux, K);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, pmL_i, K, aux, K, &beta, pm_o, M);
    }
    hipFreeAsync(aux, hipStreamDefault);
}
void MathGPU::MatMulTT(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, int M, int K, int N)
{
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_T, M, N, K, &alpha, pmL_i, K, pmR_i, N, &beta, pm_o, M);
}
void MathGPU::MatMulTWT(double beta, double *pm_o, double alpha, double *pmL_i, double *pmR_i, double *pw_i, int M, int K, int N)
{
    double *aux;
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipMallocAsync(&aux, sizeof(double) * min(M, N) * K, hipStreamDefault);
    Zero(aux, min(M, N) * K);
    if (M < N)
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_LEFT, K, M, pmL_i, K, pw_i, 1, aux, K);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_N, M, N, K, &alpha, aux, K, pmR_i, N, &beta, pm_o, M);
    }
    else
    {
        hipblasDdgmm(cublasHandle, hipblasSideMode_t::HIPBLAS_SIDE_RIGHT, N, K, pmR_i, M, pw_i, 1, aux, M);
        hipblasDgemm(cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_T, M, N, K, &alpha, pmL_i, K, aux, N, &beta, pm_o, M);
    }
    hipFreeAsync(aux, hipStreamDefault);
}
void MathGPU::Mean(double *pv_o, double *pm_i, int lengthI, int lengthJ)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(lengthI, T.x));
    CUDA_Mean<<<B, T, 0, hipStreamDefault>>>(pv_o, pm_i, lengthI, lengthJ);
}
void MathGPU::Mean(double *pv_o, double *pm_i, double *pw_i, int lengthI, int lengthJ)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(lengthI, T.x));
    CUDA_Mean<<<B, T, 0, hipStreamDefault>>>(pv_o, pm_i, pw_i, lengthI, lengthJ);
}
bool MathGPU::Compare(double *pvL_i, double *pvR_i, int length)
{
    dim3 T(THREAD_COUNT);
    dim3 B(CEIL(length, T.x));
    double *pDev;
    double res = -1.0;
    hipblasSetStream(cublasHandle, hipStreamDefault);
    hipMallocAsync(&pDev, sizeof(double) * length, hipStreamDefault);
    CUDA_Sub<<<B, T, 0, hipStreamDefault>>>(pDev, pvL_i, pvR_i, length);
    hipblasDnrm2(cublasHandle, length, pDev, 1, &res);
    hipFreeAsync(pDev, hipStreamDefault);
    return true;
}
void MathGPU::Diag(double *pv_o, double *pm_i, int length)
{
    hipblasDcopy(cublasHandle, length, pm_i, length + 1, pv_o, 1);
}
void MathGPU::CholeskyDecomposition(double *pm_o, double *pm_i, int length)
{
    int size = 0;
    int *pdInfo;
    double *pdAux;
    hipsolverSetStream(cusolverDnHandle, hipStreamDefault);
    hipMemcpyAsync(pm_o, pm_i, sizeof(double) * length * length, hipMemcpyKind::hipMemcpyDeviceToDevice, hipStreamDefault);
    hipsolverDnDpotrf_bufferSize(cusolverDnHandle, hipblasFillMode_t::HIPBLAS_FILL_MODE_LOWER, length, pm_o, length, &size);
    hipMallocAsync(&pdInfo, sizeof(int), hipStreamDefault);
    hipMallocAsync(&pdAux, sizeof(double) * size, hipStreamDefault);
    hipsolverDnDpotrf(cusolverDnHandle, hipblasFillMode_t::HIPBLAS_FILL_MODE_LOWER, length, pm_o, length, pdAux, size, pdInfo);
    hipFreeAsync(pdAux, hipStreamDefault);
    hipFreeAsync(pdInfo, hipStreamDefault);
}
void MathGPU::CholeskySolver(double *pm_o, double *pmL_i, double *pmR_i, int M, int K, int N)
{
    if (M != K)
    {
        return;
    }
    double *pm;
    int *pdInfo;
    hipsolverSetStream(cusolverDnHandle, hipStreamDefault);
    hipMallocAsync(&pm, sizeof(double) * M * K, hipStreamDefault);
    hipMallocAsync(&pdInfo, sizeof(int), hipStreamDefault);
    CholeskyDecomposition(pm, pmL_i, K);
    hipMemcpyAsync(pm_o, pmR_i, sizeof(double) * M * N, hipMemcpyKind::hipMemcpyDeviceToDevice, hipStreamDefault);
    hipsolverDnDpotrs(cusolverDnHandle, hipblasFillMode_t::HIPBLAS_FILL_MODE_LOWER, M, N, pm, M, pm_o, M, pdInfo);
    hipFreeAsync(pdInfo, hipStreamDefault);
    hipFreeAsync(pm, hipStreamDefault);
    hipDeviceSynchronize();
}
