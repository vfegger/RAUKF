#include "hip/hip_runtime.h"
#include "../include/hc2D.hpp"

__host__ __device__ inline double HC2D::C(double x, double y)
{
    return 385 * 8960;
}
__host__ __device__ inline double HC2D::K(double x, double y)
{
    return 400;
}

void HC2D::validate(HCParms &parms)
{
    if (!(refparms == parms))
    {
        refparms = parms;
        if (AI.host())
        {
            AI.free();
        }
        int Lxy = parms.Lx * parms.Ly;
        int Lu = 1 + 2 * (parms.Lx + parms.Ly);
        int L = Lxy + Lxy;
        int L2 = L * L;
        AI.alloc(L2);
        BE.alloc(L2);
        CE.alloc(L * Lu);
        ATA.alloc(L2);
        JX.alloc(L2);
        JU.alloc(L * Lu);
        isValid = false;
    }
}

void HC2D::CPU::ImplicitScheme(HCParms &parms, int strideTQ, int strideAC)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1 + 2 * (parms.Lx + parms.Ly);
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;
    double gamma = parms.gamma;

    double *pAI = AI.host();
    double *pBE = BE.host();
    double *pCE = CE.host();
    double *pATA = ATA.host();
    double *paux = (double *)malloc(sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ, *pTcT, *pTcQ;
    MathCPU::Zero(pAI, L * L);
    MathCPU::Identity(pBE, L, L);
    MathCPU::Zero(pCE, L * Lu);
    MathCPU::Zero(pATA, L * L);
    MathCPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, strideTQ);
    pTcT = pCE + std::max(0, strideAC) * L + std::max(0, -strideTQ);
    pTcQ = pCE + std::max(0, strideAC) * L + std::max(0, strideTQ);

    double *JXh = JX.host();
    double *JUh = JU.host();
    // Difusion Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            double aux = 0.0;
            if (i != 0)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] = -dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            }
            if (i != Lx - 1)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] = -dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            }
            if (j != 0)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] = -dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            }
            if (j != Ly - 1)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] = -dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            }
            pTT[index * L + index] = 1.0 + aux + dt * h / (c * CT);
        }
    }

    // Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQT[index * L + index] = -dt * amp / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQQ[index * L + index] = 1.0;
        }
    }

    // Ambient Temperature Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pTaT[index] = h / (c * CT);
        }
    }
    // Contour Temperature Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        double KT0 = K((0.0) * dx, (j + 0.5) * dy);
        double KT1 = K((Lx)*dx, (j + 0.5) * dy);
        double CT0 = C((0.0) * dx, (j + 0.5) * dy);
        double CT1 = C((Lx)*dx, (j + 0.5) * dy);
        int index0 = j * Lx + 0;
        int index1 = j * Lx + Lx - 1;
        pTcT[j * L + index0] = -dt * gamma * KT0 / (CT0 * dx * dx);
        pTcT[(Ly + j) * L + index1] = -dt * gamma * KT1 / (CT1 * dx * dx);
    }
    for (int i = 0; i < Lx; ++i)
    {
        double KT0 = K((i + 0.5) * dx, (0.0) * dy);
        double KT1 = K((i + 0.5) * dx, (Ly)*dy);
        double CT0 = C((i + 0.5) * dx, (0.0) * dy);
        double CT1 = C((i + 0.5) * dx, (Ly)*dy);
        int index0 = 0 * Lx + i;
        int index1 = (Ly - 1) * Lx + i;
        pTcT[(2 * Ly + i) * L + index0] = -dt * gamma * KT0 / (CT0 * dy * dy);
        pTcT[(2 * Ly + Lx + i) * L + index1] = -dt * gamma * KT1 / (CT1 * dy * dy);
    }

    MathCPU::MatMulTN(0.0, pATA, 1.0, pAI, pAI, L, L, L);
    MathCPU::MatMulTN(0.0, paux, 1.0, pAI, pBE, L, L, L);
    // Solve JX = (A^T * A)^-1 * A^T * B
    MathCPU::CholeskySolver(JXh, pATA, paux, L, L, L);

    MathCPU::MatMulTN(0.0, paux, 1.0, pAI, pCE, L, L, Lu);
    // Solve JU = (A^T * A)^-1 * A^T * C
    MathCPU::CholeskySolver(JUh, pATA, paux, L, L, Lu);
    free(paux);
}

void HC2D::CPU::ExplicitScheme(HCParms &parms, int strideTQ, int strideAC)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1 + 2 * (parms.Lx + parms.Ly);
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;
    double gamma = parms.gamma;

    double *pAI = AI.host();
    double *pBE = BE.host();
    double *pCE = CE.host();
    double *pATA = ATA.host();
    double *paux = (double *)malloc(sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ, *pTcT, *pTcQ;
    MathCPU::Identity(pAI, L, L);
    MathCPU::Zero(pBE, L * L);
    MathCPU::Zero(pCE, L * Lu);
    MathCPU::Zero(pATA, L * L);
    MathCPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, strideTQ);
    pTcT = pCE + std::max(0, strideAC) * L + std::max(0, -strideTQ);
    pTcQ = pCE + std::max(0, strideAC) * L + std::max(0, strideTQ);

    double *JXh = JX.host();
    double *JUh = JU.host();
    // Difusion Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            double aux = 0.0;
            if (i != 0)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] = dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            }
            if (i != Lx - 1)
            {
                aux += dt * KT / (CT * dx * dx);
                pTT[(index + 1) * L + index] = dt * KT / (CT * dx * dx);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
                pTT[(index - 1) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            }
            if (j != 0)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] = dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            }
            if (j != Ly - 1)
            {
                aux += dt * KT / (CT * dy * dy);
                pTT[(index + Lx) * L + index] = dt * KT / (CT * dy * dy);
            }
            else
            {
                aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
                pTT[(index - Lx) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            }
            pTT[index * L + index] = 1.0 - aux - dt * h / (c * CT);
        }
    }

    // Heat Flux Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQT[index * L + index] = dt * amp / (c * CT);
        }
    }
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pQQ[index * L + index] = 1.0;
        }
    }

    // Ambient Temperature Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double CT = C((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pTaT[index] = h / (c * CT);
        }
    }
    // Contour Temperature Temperature Contribution
    for (int j = 0; j < Ly; ++j)
    {
        double KT0 = K((0.0) * dx, (j + 0.5) * dy);
        double KT1 = K((Lx)*dx, (j + 0.5) * dy);
        double CT0 = C((0.0) * dx, (j + 0.5) * dy);
        double CT1 = C((Lx)*dx, (j + 0.5) * dy);
        int index0 = j * Lx + 0;
        int index1 = j * Lx + Lx - 1;
        pTcT[j * L + index0] = -dt * gamma * KT0 / (CT0 * dx * dx);
        pTcT[(Ly + j) * L + index1] = -dt * gamma * KT1 / (CT1 * dx * dx);
    }
    for (int i = 0; i < Lx; ++i)
    {
        double KT0 = K((i + 0.5) * dx, (0.0) * dy);
        double KT1 = K((i + 0.5) * dx, (Ly)*dy);
        double CT0 = C((i + 0.5) * dx, (0.0) * dy);
        double CT1 = C((i + 0.5) * dx, (Ly)*dy);
        int index0 = 0 * Lx + i;
        int index1 = (Ly - 1) * Lx + i;
        pTcT[(2 * Ly + i) * L + index0] = -dt * gamma * KT0 / (CT0 * dy * dy);
        pTcT[(2 * Ly + Lx + i) * L + index1] = -dt * gamma * KT1 / (CT1 * dy * dy);
    }

    // Solve JX = (A^T * A)^-1 * A^T * B
    MathCPU::Copy(JXh, pBE, L * L);

    // Solve JU = (A^T * A)^-1 * A^T * C
    MathCPU::Copy(JUh, pCE, L * Lu);
}

void HC2D::CPU::EvolutionMatrix(HCParms &parms, double *pmXX_o, double *pmUX_o, int strideTQ, int strideAC)
{
    validate(parms);
    if (!isValid)
    {
#if IMPLICIT_SCHEME == 1
        ImplicitScheme(parms, strideTQ, strideAC);
#else
        ExplicitScheme(parms, strideTQ, strideAC);
#endif
        isValid = true;
    }
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    MathCPU::Copy(pmXX_o, JX.host(), L * L);
    MathCPU::Copy(pmUX_o, JU.host(), L * Lu);
}

void HC2D::CPU::EvaluationMatrix(HCParms &parms, double *pmH_o, int strideTQ)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    double dx = parms.dx;
    double dy = parms.dy;
    double c = parms.Sz;
    double amp = parms.amp;

    double *pmTT, *pmQT;
    pmTT = pmH_o + std::max(-strideTQ, 0) * Lxy;
    pmQT = pmH_o + std::max(strideTQ, 0) * Lxy;
    // Surface Temperature
    for (int j = 0; j < Ly; j++)
    {
        for (int i = 0; i < Lx; ++i)
        {
            double KT = K((i + 0.5) * dx, (j + 0.5) * dy);
            index = j * Lx + i;
            pmTT[index * Lxy + index] = 1.0;
#if ILSA == 1
            pmQT[index * Lxy + index] = -c * amp / (6.0 * KT);
#endif
        }
    }
}

__global__ void ImplicitScheme_A(double *pmTT, double *pmTQ, double *pmQT, double *pmQQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        double aux = 0.0;
        if (i != 0)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] = -dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (i != Lx - 1)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] = -dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (j != 0)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] = -dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
        }
        if (j != Ly - 1)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] = -dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += -(1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
        }
        pmTT[index * L + index] = 1.0 + aux + dt * h / (c * CT);
        pmQT[index * L + index] = -dt * amp / (c * CT);
        pmQQ[index * L + index] = 1.0;
    }
}

__global__ void ImplicitScheme_C(double *pmTaT, double *pmTaQ, double *pmTcT, double *pmTcQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        pmTaT[index] = dt * h / (c * CT);
        if (i == 0)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmTcT[k * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (i == Lx - 1)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmTcT[(k + Ly) * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (j == 0)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmTcT[(k + 2 * Ly) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
        if (j == Ly - 1)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmTcT[(k + Lx + 2 * Ly) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
    }
}

__global__ void ExplicitScheme_B(double *pmTT, double *pmTQ, double *pmQT, double *pmQQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        double aux = 0.0;
        if (i != 0)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index - 1) * L + index] = dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (i != Lx - 1)
        {
            aux += dt * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] = dt * KT / (CT * dx * dx);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
            pmTT[(index + 1) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dx * dx);
        }
        if (j != 0)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index - Lx) * L + index] = dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
        }
        if (j != Ly - 1)
        {
            aux += dt * KT / (CT * dy * dy);
            pmTT[(index + Lx) * L + index] = dt * KT / (CT * dy * dy);
        }
        else
        {
            aux += (3.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
            pmTT[(index + 1) * L + index] += (1.0 / 2.0) * dt * gamma * KT / (CT * dy * dy);
        }
        pmTT[index * L + index] = 1.0 - aux - dt * h / (c * CT);
        pmQT[index * L + index] = dt * amp / (c * CT);
        pmQQ[index * L + index] = 1.0;
    }
}

__global__ void ExplicitScheme_C(double *pmTaT, double *pmTaQ, double *pmTcT, double *pmTcQ, int Lx, int Ly, double dx, double dy, double dt, double c, double amp, double h, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int Lxy = Lx * Ly;
    int L = Lxy + Lxy;
    int index = j * Lx + i;
    if (i < Lx && j < Ly)
    {
        double KT = HC2D::K((i + 0.5) * dx, (j + 0.5) * dy);
        double CT = HC2D::C((i + 0.5) * dx, (j + 0.5) * dy);
        pmTaT[index] = dt * h / (c * CT);
        if (i == 0)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmTcT[k * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (i == Lx - 1)
        {
            for (int k = 0; k < Ly; k++)
            {
                pmTcT[(k + Ly) * L + index] = -dt * gamma * KT / (CT * dx * dx);
            }
        }
        if (j == 0)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmTcT[(k + 2 * Ly) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
        if (j == Ly - 1)
        {
            for (int k = 0; k < Lx; k++)
            {
                pmTcT[(k + Lx + 2 * Ly) * L + index] = -dt * gamma * KT / (CT * dy * dy);
            }
        }
    }
}

void HC2D::GPU::ImplicitScheme(HCParms &parms, int strideTQ, int strideAC)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;
    double gamma = parms.gamma;

    double *pAI = AI.dev();
    double *pBE = BE.dev();
    double *pCE = CE.dev();
    double *pATA = ATA.dev();
    double *paux = NULL;
    hipMalloc(&paux, sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ, *pTcT, *pTcQ;
    MathGPU::Zero(pAI, L * L);
    MathGPU::Identity(pBE, L, L);
    MathGPU::Zero(pCE, L * Lu);
    MathGPU::Zero(pATA, L * L);
    MathGPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, strideTQ);
    pTcT = pCE + std::max(0, strideAC) * L + std::max(0, -strideTQ);
    pTcQ = pCE + std::max(0, strideAC) * L + std::max(0, strideTQ);

    double *JXh = JX.dev();
    double *JUh = JU.dev();

    dim3 T(16, 16);
    dim3 B((L + T.x - 1) / T.x, (L + T.y - 1) / T.y);
    ImplicitScheme_A<<<B, T>>>(pTT, pTQ, pQT, pQQ, Lx, Ly, dx, dy, dt, c, amp, h, gamma);
    MathGPU::Identity(pBE, L, L);
    ImplicitScheme_C<<<B, T>>>(pTaT, pTaQ, pTcT, pTcQ, Lx, Ly, dx, dy, dt, c, amp, h, gamma);

    MathGPU::MatMulTN(0.0, pATA, 1.0, pAI, pAI, L, L, L);
    MathGPU::MatMulTN(0.0, paux, 1.0, pAI, pBE, L, L, L);
    // Solve JX = (A^T * A)^-1 * A^T * B
    MathGPU::CholeskySolver(JXh, pATA, paux, L, L, L);

    MathGPU::MatMulTN(0.0, paux, 1.0, pAI, pCE, L, L, Lu);
    // Solve JU = (A^T * A)^-1 * A^T * C
    MathGPU::CholeskySolver(JUh, pATA, paux, L, L, Lu);
    hipFree(paux);
}

void HC2D::GPU::ExplicitScheme(HCParms &parms, int strideTQ, int strideAC)
{
    int index;
    int Lx = parms.Lx;
    int Ly = parms.Ly;
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    double dx = parms.dx;
    double dy = parms.dy;
    double dt = parms.dt;
    double c = parms.Sz;
    double amp = parms.amp;
    double h = parms.h;
    double gamma = parms.gamma;

    double *pAI = AI.dev();
    double *pBE = BE.dev();
    double *pCE = CE.dev();
    double *pATA = ATA.dev();
    double *paux = NULL;
    hipMalloc(&paux, sizeof(double) * L * L);
    double *pTT, *pTQ, *pQT, *pQQ, *pTaT, *pTaQ, *pTcT, *pTcQ;
    MathGPU::Identity(pAI, L, L);
    MathGPU::Zero(pBE, L * L);
    MathGPU::Zero(pCE, L * Lu);
    MathGPU::Zero(pATA, L * L);
    MathGPU::Zero(paux, L * L);
    pTT = pAI + std::max(0, -strideTQ) * (L + 1);
    pQQ = pAI + std::max(0, strideTQ) * (L + 1);
    pTQ = pTT + strideTQ;
    pQT = pQQ - strideTQ;
    pTaT = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, -strideTQ);
    pTaQ = pCE + std::max(0, -strideAC) * (2 * (parms.Lx + parms.Ly)) * L + std::max(0, strideTQ);
    pTcT = pCE + std::max(0, strideAC) * L + std::max(0, -strideTQ);
    pTcQ = pCE + std::max(0, strideAC) * L + std::max(0, strideTQ);

    double *JXh = JX.dev();
    double *JUh = JU.dev();

    dim3 T(16, 16);
    dim3 B((L + T.x - 1) / T.x, (L + T.y - 1) / T.y);
    ImplicitScheme_A<<<B, T>>>(pTT, pTQ, pQT, pQQ, Lx, Ly, dx, dy, dt, c, amp, h, gamma);
    MathGPU::Identity(pBE, L, L);
    ImplicitScheme_C<<<B, T>>>(pTaT, pTaQ, pTcT, pTcQ, Lx, Ly, dx, dy, dt, c, amp, h, gamma);

    // Solve JX = (A^T * A)^-1 * A^T * B
    MathGPU::Copy(JXh, pBE, L * L);

    // Solve JU = (A^T * A)^-1 * A^T * C
    MathGPU::Copy(JUh, pCE, L * Lu);
}

void HC2D::GPU::EvolutionMatrix(HCParms &parms, double *pmXX_o, double *pmUX_o, int strideTQ, int strideAC)
{
    validate(parms);
    if (!isValid)
    {
#if IMPLICIT_SCHEME == 1
        ImplicitScheme(parms, strideTQ, strideAC);
#else
        ExplicitScheme(parms, strideTQ, strideAC);
#endif
        std::cout << "Saving matrices for Evolution" << std::endl;
        isValid = true;
    }
    int Lxy = parms.Lx * parms.Ly;
    int Lu = 1;
    int L = Lxy + Lxy;
    MathGPU::Copy(pmXX_o, JX.dev(), L * L);
    MathGPU::Copy(pmUX_o, JU.dev(), L * Lu);
}

void HC2D::GPU::EvaluationMatrix(HCParms &parms, double *pmH_o, int strideTQ)
{
    int Lxy = parms.Lx * parms.Ly;

    double *pm;

    pm = (double *)malloc(sizeof(double) * 2 * Lxy * Lxy);
    for (int i = 0; i < 2 * Lxy * Lxy; i++)
    {
        pm[i] = 0.0;
    }

    HC2D::CPU::EvaluationMatrix(parms, pm, strideTQ);

    hipMemcpy(pmH_o, pm, 2 * sizeof(double) * Lxy * Lxy, hipMemcpyKind::hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    free(pm);
}